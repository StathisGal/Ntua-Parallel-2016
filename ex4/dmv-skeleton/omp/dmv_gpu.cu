#include "hip/hip_runtime.h"
/*
 *  dmv_gpu.cu -- Template for DMV GPU kernels
 *
 *  Copyright (C) 2010-2013, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2010-2013, Vasileios Karakasis
 */ 
#include <stdio.h>
#include "dmv.h"


/*
 *  Utility function to get the thread ID within the
 *  global working space.
 */ 
__device__ int get_global_tid()
{
    return (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x*blockDim.y +
        blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Utility function to get the thread ID within the
 *  local/block working space.
 */ 
__device__ int get_local_tid()
{
    return blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Naive kernel
 */ 
__global__ void dmv_gpu_naive (const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
    /*
     * FILLME: fill the code for the naive kernel.
     */ 
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i=0; i<n; i++)
	  {
	    if(j<n)
		y[j] += a[i+j*n]*x[i];
	  }
}

/*
 *  Coalesced memory acceses
 */
__global__ void dmv_gpu_coalesced(const value_t *a, const value_t *x,
                                  value_t *y, size_t n)
{
  /*
   * FILLME: fill the code for the coalesced kernel.
   */
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  float tmp=0;
  for (int i=0; i<n*n; i+=n)
    {
      /* if(j==0) */
      /* printf("i:%d j:%d i/n:%d a:%f x:%f y:%f",i,j,i/n,a[i+j],x[i/n],y[j]); */
      if(j<n) tmp += a[i+j]*x[i/n];
    } 
  y[j] = tmp;
}

/*
 *  Use of shared memory
 */
__global__ void dmv_gpu_shmem(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{

  int block_width, block_height; 
  if (  n % gridDim.x)
    block_height = (n / gridDim.x) + 1;
    else
      block_height = (n / gridDim.x);
  if ( n % gridDim.y)
    block_width = (n / gridDim.y) + 1;
  else
    block_width = (n / gridDim.y);
  
  extern __shared__ value_t x_sh[];
  
  __shared__ int *blockElt,*block_x_Ind,*block_y_Ind;

  blockElt=(int*)&x_sh[block_width];
  block_x_Ind=(int*)&x_sh[block_width+1];
  block_y_Ind=(int*)&x_sh[block_width+2];
  
  // to y leei tin grammi kai to x tin stili
  if(threadIdx.x == 0)
    {
      /* printf("w:%d h:%d\n",block_width,block_height); */
      if((blockIdx.y+1)* block_width > n)
	{
	  /* printf("mpikeee:%d",n % block_width); */
	  *blockElt = (n % block_width);
	}
      else
	*blockElt =block_width;

      *block_x_Ind = blockIdx.x * block_height;
      *block_y_Ind = blockIdx.y * block_width;
    }
  __syncthreads();
  
  //I think it needs sync

  int thread_x_Ind =blockIdx.x * block_height + threadIdx.x;
    

  /* if (thread_x_Ind < n) */
  if(threadIdx.x <*blockElt)
    x_sh[threadIdx.x] = x[*block_y_Ind + threadIdx.x];
  /* if(threadIdx.x == 0 ) */
  /*   for(int i=0; i<*blockElt; i++) */
  /*     printf("edw exoume %d a:%f x:%f\n",*blockElt,a[(*block_y_Ind + i) * n  + *block_x_Ind + threadIdx.x],x_sh[i]); */
  __syncthreads();

  value_t cSum = 0;

  /* if(threadIdx.x == 1) */
  /*   { */
  /* /\*     printf("blockElt:%d, thread_y_Ind:%d, block_x_Ind:%d, block_y_Ind:%d\n",*blockElt,thread_y_Ind,*block_x_Ind,*block_y_Ind); *\/ */
  /*   } */
  /* printf("thread:%d blockElt:%d blockx:%d, blocky:%d\n",threadIdx.x,*blockElt,blockIdx.x,blockIdx.y); */
  if(thread_x_Ind < n     )
    {
      for(int i=0; i<*blockElt; i++)
  	{
	  /* if(threadIdx.x == 0 && *block_x_Ind == 0 && *block_y_Ind == 0) */
	  /*   printf("a:%f x:%f new:%f cSum:%f\n",a[(*block_y_Ind + i) * n  + *block_x_Ind + threadIdx.x],x_sh[i], x_sh[i]*a[(*block_y_Ind + i) * n  + *block_x_Ind + threadIdx.x],cSum); */
  	  cSum += x_sh[i]*a[(*block_y_Ind + i) * n  + *block_x_Ind + threadIdx.x];
  	}
	  /* if(threadIdx.x == 0 && *block_x_Ind == 0 && *block_y_Ind == 0) */
	    /* printf("sum is: %f\n", cSum);       */
      atomicAdd((float*)&y[thread_x_Ind],(float)cSum);
    }

}
